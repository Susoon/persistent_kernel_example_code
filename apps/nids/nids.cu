#include "hip/hip_runtime.h"
#include "nids.h"

//CKJUNG, 19.03.22 NIDS functions
__device__ int lookup2D(int* trie, int col, int row)
{ 
	if(row == -1)
		row = 0;
	return trie[row*MAXC + col];
} 

__device__ int lookup1D(int* arr, int point) 
{ 
	if(point == -1)
		point = 0; 
	return arr[point];
} 

__global__ void testtt(void) {
	START_YLW
	printf("_______________[%s] HELLO!!!__________________\n", __FUNCTION__);
	END
}

// CKJUNG, 19.01.30 [NF#3:NIDS]------------------------------------- 
__global__ void nids(struct mempool** mempool, uint32_t* pkt_cnt, int** d_dstTrie, int** d_dstFailure, int** d_dstOutput, struct portGroup *d_pg)
{ 
	int i;
	__shared__ unsigned char xlatcase[256];

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int pktid = tid / THD_PER_PKT;
	int dataid = tid % THD_PER_PKT;

	struct pkt_buf* buf = NULL;
	__shared__ struct pkt_buf* buf_pool[512];
	struct mempool* mini_mempool = NULL;
	if(pktid < 512)
		mini_mempool = mempool[pktid];

	if(threadIdx.x == 0){
		for(i = 0; i < 256; i++)
			xlatcase[i] = (unsigned char)TOUPPER(i); // Init xlatcase : Convert Lower to Upper
		for(int i = 0; i < 512; i++)
			buf_pool[i] = NULL;
	}

#if 0 

	if(threadIdx.x == NF_T_NUM - 1){
		START_RED
		printf("[%s] threadIdx.x %d is alive!\n", __FUNCTION__, threadIdx.x);
		END
	}
#endif

	
	while(true) { //Persistent Kernel
		__syncthreads();
		if(pktid < 512){
			if(dataid == 0){
				buf_pool[pktid] = pkt_buf_alloc(mini_mempool);
				//buf_pool[pktid] = pkt_buf_extract(mini_mempool, 1);
            }

			__syncthreads();
			buf = buf_pool[pktid];
			if(buf != NULL){
#if 1 
				struct udphdr* udph = (struct udphdr *)(buf->data + sizeof(struct ethhdr) + sizeof(struct iphdr));

				// Extract "portNUM" & "Length of payload"
				int dst_port = NTOHS(udph->dest);
				//int payload_len = (int)NTOHS(udph->len) - sizeof(struct udphdr);
				int payload_len = PKT_DATA_SIZE - 42;
		
				unsigned char* payload = buf->data + sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct udphdr) + dataid*DATA_PER_THD;

				int y = 0, r, s, cnt = 0;
				r = 0;
				int ret = 0;
				int curPoint = sizeof(struct ethhdr) + sizeof(struct iphdr) + sizeof(struct udphdr) + dataid*DATA_PER_THD;

				if(d_pg->dstPortMap[dst_port] == NULL){
				}else{
					int *tmp_trie = d_dstTrie[dst_port];
					int *tmp_failure = d_dstFailure[dst_port];
					int *tmp_output = d_dstOutput[dst_port];

					while(y + curPoint < PKT_SIZE){
						if(payload[y]>='a' && payload[y]<='z') // Convert Lower to Upper
							payload[y] = xlatcase[payload[y]];

						// string matching with Trie and Failure link
						while((s = lookup2D(tmp_trie, payload[y], r)) == -1){
							r = lookup1D(tmp_failure, r);
						}
						if(s == 0){
							if(y >= payload_len/THD_PER_PKT)
								break;
						}
						r = s;
						ret = lookup1D(tmp_output, r);
						cnt += ret;
						y++;
					}				
				}
				//memset(payload, 0, 10);
				//memcpy(payload + 2, &dst_port, sizeof(int));
				//memset(payload + 6, 0, 4);

#endif
				__syncthreads();
				if(dataid == 0){
/*
					buf->app_idx = 2;
					buf = NULL;
*/
                    pkt_buf_free(&(buf));
                    atomicAdd(&pkt_cnt[1], 1);
					buf_pool[pktid] = NULL;
				}
			}
		}
	}
}
	// ~CKJUNG, ---------------------------------------------------------- 

extern "C"
void initialize_nids(struct mempool** mempool, uint32_t *pkt_cnt)
{
#if 1
	// CKJUNG, 19.03.22 [NF #3: NIDS] Setting DST TRIEs, Failures, Outputs /////////////////////////
	char buf[30]; 
	char *tok;
	int portNum; 
	int i, j;

	queue<int> q;

	printf("____[Initialize]__NF #3__NIDS__\n");
	// DRAM : TRIEs, Failures, Outputs 
	struct portGroup pg;
	memset(&pg, 0, sizeof(struct portGroup)); 
	pg.dstTrie = (int**)calloc(sizeof(int*),MAX_PORTS); 
	pg.dstFailure = (int**)calloc(sizeof(int*),MAX_PORTS); 
	pg.dstOutput = (int**)calloc(sizeof(int*),MAX_PORTS);

	// GDDR : TRIEs, Failures, Outputs 
	struct portGroup *d_pg;
	ASSERTRT(hipMalloc((void**)&d_pg, sizeof(struct portGroup))); 
	ASSERTRT(hipMemset(d_pg, 0, sizeof(struct portGroup)));

	// [TODO] 19.03.22. How to access "Double pointer in struct which is in GPU?"
	int **d_dstTrie; 
	int **d_dstFailure; 
	int **d_dstOutput;
	ASSERTRT(hipMalloc((void**)&d_dstTrie, sizeof(int*)*MAX_PORTS));
	ASSERTRT(hipMalloc((void**)&d_dstFailure, sizeof(int*)*MAX_PORTS));
	ASSERTRT(hipMalloc((void**)&d_dstOutput, sizeof(int*)*MAX_PORTS)); 

	FILE* fp = fopen("./apps/lib/ck_dst_trie.txt","r");

	while((fgets(buf, LINE_LENGTH, fp)) != NULL)
	{ 
		if(!strcmp(buf, " ")||!strcmp(buf, "\n")) 
			continue; 
		// CKJUNG, For port Num
		tok = strtok(buf, " ");
		if(!strcmp(tok, "dst")){ 
			portNum = atoi(strtok(NULL, " ")); 
		}else if(!strcmp(tok, "src")){
			portNum = atoi(strtok(NULL, " ")); 
		}else{ // Gen or After portNum
			int Depth = atoi(buf); 
			if(Depth == 0) // If meaningless then continue,,
				continue;
			// CKJUNG, Initialize Array 
			int arr[Depth][MAXC];
			for(i = 0; i < Depth; i++)
				for(j = 0; j < MAXC; j++) 
					arr[i][j] = -1; 
			// ~CKJUNG

			pg.dstOutput[portNum] = (int*)malloc(sizeof(int)*(Depth+1));
			for(i = 0; i < Depth+1; i++)
				(pg.dstOutput[portNum])[i] = 0;

			// CKJUNG, Fill the Array
			int prev = -1;
			int ptnLen = 0;
			int numPtn = 1;
			for(i = 0; i < Depth; i++)
			{
				int stateNum;
				int row, col;
				fgets(buf, LINE_LENGTH, fp);
				//printf("buf: %s\n", buf); 
				stateNum = atoi(strtok(buf, ":"));
				tok = strtok(NULL, ":");
				row = atoi(strtok(tok, " "));
				if(prev > row){ 
					ptnLen = 1;
					numPtn++;
					pg.dstOutput[portNum][stateNum-1] = 1; // Filling Output vector 1.
				}else if(i == Depth-1){
					ptnLen++;
					pg.dstOutput[portNum][stateNum] = 1; // Filling Output vector 2.
				}else{
					ptnLen++;
				}
				prev = row;
				col = atoi(strtok(NULL, " ")); 
				arr[row][col] = stateNum; 
			} 

			// 1st Row should be filled by "zeroes". 
			// Because they are root nodes
			for(i = 0; i < MAXC; i++)
				if(arr[0][i] == -1)
					arr[0][i] = 0;

			//CKJUNG, [TODO, 19.02.18 16:43] Making failure State

			// Initialize Failure link as -1
			int oo;
			pg.dstFailure[portNum] = (int*)malloc(sizeof(int)*(Depth+1)); 
			for(oo = 0; oo < Depth+1; oo++) 
				(pg.dstFailure[portNum])[oo] = -1; 

			// Initiailize Failure link of root node as 0(root)
			int ch;
			for(ch = 0; ch < MAXC; ch++)
			{ 
				// If root node has some child nodes
				if(arr[0][ch] != 0) 
				{
					(pg.dstFailure[portNum])[arr[0][ch]] = 0;
					q.push(arr[0][ch]);
				}
			} 

			while(q.size())
			{ 
				int state = q.front(); 
				if(state >= Depth)
					break;
				q.pop();
				for(ch = 0; ch < MAXC; ch++)
				{
					if(arr[state][ch] != -1)
					{
						int failure = (pg.dstFailure[portNum])[state];
						while(arr[failure][ch] == -1) 
							failure = (pg.dstFailure[portNum])[failure]; 

						failure = arr[failure][ch]; 
						(pg.dstFailure[portNum])[arr[state][ch]] = failure; 

						(pg.dstOutput[portNum])[arr[state][ch]] += (pg.dstOutput[portNum])[failure]; 
						q.push(arr[state][ch]); 
					} 
				}
			} 

			//[THINK] Every time we malloc here, we get NEW ADDRESS for each TRIE, CKJUNG 
			pg.dstTrie[portNum] = (int*)malloc(sizeof(int)*Depth*MAXC); 
			for(i = 0; i < Depth; i++) 
				for(j = 0; j < MAXC; j++) 
					pg.dstTrie[portNum][i*MAXC+j] = arr[i][j]; 

			//[THINK] We SHOULDN'T FREE "brr" until the end of the program!!, CKJUNG
			pg.dstPortMap[portNum] = 1; // Set portMap 
			pg.dstTrieDepth[portNum] = Depth; 
		} // We've read all
	}

	///////////////////////////// CKJUNG, Copy Tries to GPU /////////////////////////////////////// 
	int *tmp_trie[MAX_PORTS]; 
	int *tmp_failure[MAX_PORTS]; 
	int *tmp_output[MAX_PORTS];
	for(i = 0; i < MAX_PORTS; i++){
		if(pg.dstPortMap[i] == 1){ // If "this port" has a TRIE,
			// CKJUNG, hipMemcpy "PortMap" & "Depth" for dst
			ASSERTRT(hipMemcpy(&(d_pg->dstPortMap[i]),&(pg.dstPortMap[i]), sizeof(int), hipMemcpyHostToDevice)); 
			ASSERTRT(hipMemcpy(&(d_pg->dstTrieDepth[i]),&(pg.dstTrieDepth[i]), sizeof(int), hipMemcpyHostToDevice));

			// CKJUNG, hipMalloc "Trie" & "Failure" & "Output" for GDDR
			ASSERTRT(hipMalloc((void**)&tmp_trie[i], (pg.dstTrieDepth[i])*MAXC*sizeof(int))); 
			ASSERTRT(hipMalloc((void**)&tmp_failure[i], (pg.dstTrieDepth[i]+1)*sizeof(int))); 
			ASSERTRT(hipMalloc((void**)&tmp_output[i], (pg.dstTrieDepth[i]+1)*sizeof(int)));

			// CKJUNG, hipMemcpy "Trie" & "Failure" & Output" to GDDR 
			ASSERTRT(hipMemcpy(tmp_trie[i], pg.dstTrie[i], (pg.dstTrieDepth[i])*MAXC*sizeof(int), hipMemcpyHostToDevice)); 
			ASSERTRT(hipMemcpy(tmp_failure[i], pg.dstFailure[i], (pg.dstTrieDepth[i]+1)*sizeof(int), hipMemcpyHostToDevice));
			ASSERTRT(hipMemcpy(tmp_output[i], pg.dstOutput[i], (pg.dstTrieDepth[i]+1)*sizeof(int), hipMemcpyHostToDevice));
		}
	}
	ASSERTRT(hipMemcpy(d_dstTrie, tmp_trie, sizeof(int*)*MAX_PORTS, hipMemcpyHostToDevice));
	ASSERTRT(hipMemcpy(d_dstFailure, tmp_failure, sizeof(int*)*MAX_PORTS, hipMemcpyHostToDevice)); 
	ASSERTRT(hipMemcpy(d_dstOutput, tmp_output, sizeof(int*)*MAX_PORTS, hipMemcpyHostToDevice));

	hipStream_t cuda_stream4;
	ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream4,hipStreamNonBlocking));

	START_BLU
	printf("[NIDS] # of Thread Blocks : %d, # of Threads : %d\n", NF_TB_NUM, NF_T_NUM);
	END

	nids<<< NF_TB_NUM, NF_T_NUM, 0, cuda_stream4 >>> (mempool, pkt_cnt, d_dstTrie, d_dstFailure, d_dstOutput, d_pg);

	START_GRN
	printf("[Done]____[Initialize]__NF #3__NIDS__\n");
	printf("[NIDS] %s\n", hipGetErrorName(hipGetLastError()));
	END
	// ~ CKJUNG /////////////////////////////////////////////////////////////////////////////
#endif
}

